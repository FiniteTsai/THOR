// ==============================================================================
// This file is part of THOR.
//
//     THOR is free software : you can redistribute it and / or modify
//     it under the terms of the GNU General Public License as published by
//     the Free Software Foundation, either version 3 of the License, or
//     (at your option) any later version.
//
//     THOR is distributed in the hope that it will be useful,
//     but WITHOUT ANY WARRANTY; without even the implied warranty of
//     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
//     GNU General Public License for more details.
//
//     You find a copy of the GNU General Public License in the main
//     THOR directory under <license.txt>.If not, see
//     <http://www.gnu.org/licenses/>.
// ==============================================================================
//
//
//
//
// Description: test for reduction sum 
//
//
// Method: -
//
// Known limitations: None.
//
// Known issues: None.
//
//
// If you use this code please cite the following reference:
//
//       [1] Mendonca, J.M., Grimm, S.L., Grosheintz, L., & Heng, K., ApJ, 829, 115, 2016
//
// Current Code Owner: Joao Mendonca, EEG. joao.mendonca@csh.unibe.ch,
//                     Russell Deitrick, russell.deitrick@csh.unibe.ch
//                     Urs Schroffenegger, urs.schroffenegger@csh.unibe.ch
//
//
// History:
// Version Date       Comment
// ======= ====       =======
//
// 1.0     16/08/2017 Released version  (JM)
//
////////////////////////////////////////////////////////////////////////


#include <hip/hip_runtime.h>
#include <random>
#include <cmath>
#include <iostream>

#include <chrono>
#include <iomanip>
#include <sstream>

using std::cout;
using std::endl;
using std::abs;

const long MAX_BLOCK_SIZE = 4096;


double cpu_reduction_sum(double * d, long length)
{
    
    for (int stride = length; stride > 0; stride /= 2)
    {
        for (int i = 0; i < stride; i++)
            d[i] += d[i+stride];
    }

    return d[0];
}

double buf[MAX_BLOCK_SIZE];

template<int BLOCK_SIZE>
double cpu_sum(double *d, long length)
{
    long num_blocks = ceil(double(length)/double(2*BLOCK_SIZE));
    

    

    double out = 0.0;
    
    for (long i = 0; i < num_blocks; i++)
    {
        for (long j = 0; j < 2*BLOCK_SIZE; j++)
        {
            long idx = i*2*BLOCK_SIZE + j;
            
            if (idx < length)
                buf[j] = d[idx];
            else
                buf[j] = 0.0;
        }
        
            
        double o = cpu_reduction_sum(buf, BLOCK_SIZE);
        //    printf("%d: %g %g\n", i, o, out);
        out += o;
        
        
    }


    

    return out;
}


template<int BLOCK_SIZE>
__global__ void gpu_reduction_sum(double * d,
                       double * o,
                       long length)
{
   // temporary memory for all tiles in that thread
    __shared__ double ds_in[2*BLOCK_SIZE];  
 
    // import all the data from global memory
    int mem_offset1 = 2*(blockDim.x*blockIdx.x + threadIdx.x);

    if (mem_offset1 + 1 < length)
    {
        *((double2*)(&(ds_in[2*threadIdx.x]))) = *((double2*)(&(d[mem_offset1])));
    }

    else if  (mem_offset1 < length)
    {
        ds_in[2*threadIdx.x] = d[mem_offset1];
        ds_in[2*threadIdx.x + 1] = 0.0f;
    }
    else
    {
        ds_in[2*threadIdx.x] = 0.0f;
        ds_in[2*threadIdx.x + 1] = 0.0f;
    }
    
    
    // loop on stride and add
    for (int stride = blockDim.x; stride > 0; stride /= 2)
    {
        __syncthreads();
        if (threadIdx.x < stride)
            ds_in[threadIdx.x] += ds_in[threadIdx.x + stride];
    }
    
    __syncthreads();
    
    // copy to output
    
    if (threadIdx.x == 0)
       o[blockIdx.x] = ds_in[0];
    
}

        

template<int BLOCK_SIZE>
double gpu_sum(double *d, long length)
{
    int num_blocks = ceil(double(length)/double(2*BLOCK_SIZE));
    
    double * out_h = new double[num_blocks];
    double * out_d;
    double * in_d;

    //printf("num_blocks: %d\n", num_blocks);

    hipMalloc((void **)&out_d, num_blocks *     sizeof(double));
    hipMalloc((void **)&in_d , length *     sizeof(double));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Malloc: %s\n", hipGetErrorString(err));    


    hipMemcpy(in_d, d, length*sizeof(double), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("cpyH2D: %s\n", hipGetErrorString(err));    

    gpu_reduction_sum<BLOCK_SIZE><<<num_blocks, BLOCK_SIZE>>>(in_d, out_d, length);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("krnl: %s\n", hipGetErrorString(err));    

    hipMemcpy(out_h, out_d, num_blocks*sizeof(double), hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("cpyD2H: %s\n", hipGetErrorString(err));    

    double out = 0.0;
    for (int i = 0; i < num_blocks; i++)
    {
	// printf("%d: %g\n", i, out_h[i]);
        out += out_h[i];
    }
    hipFree(in_d);
    hipFree(out_d);
    delete[] out_h;
    
    
    return out;
}

template<int BLOCK_SIZE>
bool cpu_gpu_test(double * s, long size)
{
    bool overall_result = true;
    
    for (    long compute_size = size; compute_size > 0; compute_size /= 2)
    {
        std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
        double reduction_sum_CPU = cpu_sum<BLOCK_SIZE>(s, compute_size);
        std::chrono::system_clock::time_point stop = std::chrono::system_clock::now();
        auto duration_cpu = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        
        start = std::chrono::system_clock::now();
        double output_val = gpu_sum<BLOCK_SIZE>(s, compute_size);
        stop = std::chrono::system_clock::now();
        auto duration_gpu = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);       
        
        
        double output_ref = reduction_sum_CPU;
        
        bool result = output_val == output_ref;
        overall_result &= result;
        
        printf("[%ld] [%s] Computed in: CPU: %ld us, GPU: %ld us, CPU/GPU ratio: %f\n",
               compute_size,
               result?"SUCCESS":"FAIL",
               duration_cpu.count(),
               duration_gpu.count(),
               double(duration_cpu.count())/double(duration_gpu.count())
            );
        
        if (!result)
        {
            
            printf("CPU reduction sum: %32.15f\n", reduction_sum_CPU);
            printf("GPU reduction sum: %32.15f\n", output_val);
        }
    }

    return overall_result;
    
}


int main ()
{
//    long size = 500000000;
    long size = 1000000000;
//    int size = 434567890;
    
    // allocate on heap
    double * s =  new double[size];
    
    std::random_device rd;  //Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); //Standard mersenne_twister_engine seeded with rd()
    std::uniform_real_distribution<> dis(0.0, 1.0);

    
    double lin_sum = 0.0;
    printf("Generating test data\n");
    
    for (int i = 0; i < size; i++)
    {
        s[i] = dis(gen);
        //s[i] = 1.0;
        
        lin_sum += s[i];
    }

    
    printf("Computing sum over %ld elements\n", size);
    
    printf("Linear sum: %32.15f\n", lin_sum);

    bool overall_result = true;
    
    printf("\n");
    printf("Test BLOCK_SIZE = 512\n");
    printf("\n");
    overall_result &= cpu_gpu_test<512>(s, size);

    printf("\n");
    printf("Test BLOCK_SIZE = 1024\n");
    printf("\n");
    overall_result &= cpu_gpu_test<1024>(s, size);

    //bool result = abs(output_val - output_ref) < epsilon;
    
    
    if (overall_result)
        cout << "reduce sum compare SUCCESS" << endl;
    else
    {
        cout << "reduce sum compare FAIL" << endl;
    }
    
    
    delete[] s;

    
    exit(0);
}
