#include "hip/hip_runtime.h"
#include "phy_modules.h"
#include <math.h>

// Rad Trans options
double Tstar            = 4520;       // Star effective temperature [K]
double planet_star_dist = 0.015;      // Planet-star distance [au]
double radius_star      = 0.667;      // Star radius [Rsun]
double diff_fac         = 0.5;        // Diffusivity factor: 0.5-1.0
double Tlow             = 970;        // Lower boundary temperature: upward flux coming from the planet's interior
double albedo           = 0.18;       // Bond albedo
double tausw            = 532.0;      // Absorption coefficient for the shortwaves
double taulw            = 1064.0;     // Absorption coefficient for the longwaves

//  Arrays used in RT code
double *fnet_up_d     ;
double *fnet_dn_d     ;
double *tau_d         ;

//  These arrays are for temporary usage in RT code
double *dtemp         ;
double *phtemp        ;
double *ttemp         ;
double *thtemp        ;

RTSetup(double Tstar_           ,
        double planet_star_dist_,
        double radius_star_     ,
        double diff_fac_        ,
        double Tlow_            ,
        double albedo_          ,
        double tausw_           ,
        double taulw_           ) {
    
   double bc = 5.677036E-8; // Stefan–Boltzmann constant [W m−2 K−4]

   Tstar = Tstar_;
   planet_star_dist = planet_star_dist_*149597870.7;
   radius_star = radius_star_*695508;
   diff_fac = diff_fac_;
   Tlow = Tlow_;
   albedo = albedo_;
   tausw = tausw_;
   taulw = taulw_;
   double resc_flx = pow(radius_star/planet_star_dist,2.0);
   incflx = resc_flx*bc*Tstar*Tstar*Tstar*Tstar;
}


bool phy_modules_init_mem()
{
//  Rad Transfer
    hipMalloc((void **)&fnet_up_d   , nvi * point_num *     sizeof(double));
    hipMalloc((void **)&fnet_dn_d   , nvi * point_num *     sizeof(double));
    hipMalloc((void **)&tau_d       , nv * point_num * 2 *  sizeof(double));

    hipMalloc((void **)&phtemp      , nvi * point_num *     sizeof(double));
    hipMalloc((void **)&thtemp      , nvi * point_num *     sizeof(double));
    hipMalloc((void **)&ttemp       , nv * point_num *     sizeof(double));
    hipMalloc((void **)&dtemp       , nv * point_num *     sizeof(double));

    
    return true;
}

bool phy_module_init_data()
{

    return true;
}

bool phy_modules_generate_config(config_file & config_reader)
{

    config_reader.append_config_var("Tstar", Tstar, Tstar);
    config_reader.append_config_var("planet_star_dist", planet_star_dist, planet_star_dist);
    config_reader.append_config_var("radius_star", radius_star, radius_star);
    config_reader.append_config_var("diff_fac", diff_fac, diff_fac);
    config_reader.append_config_var("Tlow", Tlow, Tlow);
    config_reader.append_config_var("albedo", albedo, albedo);
    config_reader.append_config_var("tausw", tausw, tausw);
    config_reader.append_config_var("taulw", taulw, taulw);
    return true;
}

bool phy_modules_mainloop()
{

    return true;
}

bool phy_modules_store()
{

    return true;
}


bool phy_modules_free_mem()
{
    hipFree(fnet_up_d);
    hipFree(fnet_dn_d);
    hipFree(tau_d);

    hipFree(phtemp);
    hipFree(thtemp);
    hipFree(ttemp );
    hipFree(dtemp);

    
}
