#include "hip/hip_runtime.h"
// ==============================================================================
// This file is part of THOR.
//
//     THOR is free software : you can redistribute it and / or modify
//     it under the terms of the GNU General Public License as published by
//     the Free Software Foundation, either version 3 of the License, or
//     (at your option) any later version.
//
//     THOR is distributed in the hope that it will be useful,
//     but WITHOUT ANY WARRANTY; without even the implied warranty of
//     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
//     GNU General Public License for more details.
//
//     You find a copy of the GNU General Public License in the main
//     THOR directory under <license.txt>.If not, see
//     <http://www.gnu.org/licenses/>.
// ==============================================================================
//
// Build the class ESP (Exoclimes Simulation Platform)
//
//
// Description:
//   Declare and initialize variables in the model
//
// Method: -
//
//
// Known limitations: None.
//
//
// Known issues: None.
//
//
// If you use this code please cite the following reference:
//
//       [1] Mendonca, J.M., Grimm, S.L., Grosheintz, L., & Heng, K., ApJ, 829, 115, 2016
//
// Current Code Owner: Joao Mendonca, EEG. joao.mendonca@csh.unibe.ch
//
// History:
// Version Date       Comment
// ======= ====       =======
//
// 1.0     16/08/2017 Released version  (JM)
//
////////////////////////////////////////////////////////////////////////

#include "../headers/phy/chemistry_host.h"
#include "../headers/phy/valkyrie_conservation.h"
#include "../headers/phy/valkyrie_jet_steadystate.h"
#include "directories.h"
#include "esp.h"
#include "hdf5.h"
#include "storage.h"
#include <map>
#include <stdio.h>

// physical modules
#include "phy_modules.h"

__host__ ESP::ESP(int *           point_local_,
                  int *           maps_,
                  double *        lonlat_,
                  double *        Altitude_,
                  double *        Altitudeh_,
                  double *        nvecoa_,
                  double *        nvecti_,
                  double *        nvecte_,
                  double *        areasT_,
                  double *        areasTr_,
                  double *        div_,
                  double *        grad_,
                  double *        func_r_,
                  int             nl_region_,
                  int             nr_,
                  int             nv_,
                  int             nvi_,
                  int             glevel_,
                  bool            spring_dynamics_,
                  double          spring_beta_,
                  int             nlat_,
                  int             ntr_,
                  int *           zonal_mean_tab,
                  double          Rv_sponge_,
                  double          ns_sponge_,
                  double          t_shrink_,
                  int             point_num_,
                  bool            conservation,
                  benchmark_types core_benchmark_,
                  log_writer &    logwriter_) :
    nl_region(nl_region_),
    nr(nr_),
    point_num(point_num_),
    nv(nv_),
    nvi(nvi_),
    nlat(nlat_),
    ntr(ntr_),
    glevel(glevel_),
    spring_dynamics(spring_dynamics_),
    spring_beta(spring_beta_),
    logwriter(logwriter_),
    core_benchmark(core_benchmark_) {

    point_local_h = point_local_;
    maps_h        = maps_;

    lonlat_h = lonlat_;

    Altitude_h  = Altitude_;
    Altitudeh_h = Altitudeh_;

    nvecoa_h  = nvecoa_;
    nvecti_h  = nvecti_;
    nvecte_h  = nvecte_;
    areasTr_h = areasTr_;
    areasT_h  = areasT_;

    div_h  = div_;
    grad_h = grad_;

    func_r_h = func_r_;

    zonal_mean_tab_h = zonal_mean_tab;

    Rv_sponge = Rv_sponge_;
    ns_sponge = ns_sponge_;
    t_shrink  = t_shrink_;

    //
    //  Allocate Data
    if (core_benchmark != NO_BENCHMARK)
        alloc_data(conservation);
}

__host__ void ESP::alloc_data(bool conservation) {


    //
    //  Description:
    //
    //  Allocate data on host and device.
    //
    //  Allocate data in host
    //  Diagnostics
    Rho_h         = (double *)malloc(nv * point_num * sizeof(double));
    pressure_h    = (double *)malloc(nv * point_num * sizeof(double));
    temperature_h = (double *)malloc(nv * point_num * sizeof(double));
    Mh_h          = (double *)malloc(nv * point_num * 3 * sizeof(double));
    W_h           = (double *)malloc(nv * point_num * sizeof(double));
    Wh_h          = (double *)malloc(nvi * point_num * sizeof(double));

    if (conservation == true) {
        Etotal_h  = (double *)malloc(nv * point_num * sizeof(double));
        Mass_h    = (double *)malloc(nv * point_num * sizeof(double));
        AngMomx_h = (double *)malloc(nv * point_num * sizeof(double));
        AngMomy_h = (double *)malloc(nv * point_num * sizeof(double));
        AngMomz_h = (double *)malloc(nv * point_num * sizeof(double));
    }

    coeq_h  = (double *)malloc(7425 * sizeof(double));
    co2eq_h = (double *)malloc(7425 * sizeof(double));
    ch4eq_h = (double *)malloc(7425 * sizeof(double));
    h2oeq_h = (double *)malloc(7425 * sizeof(double));
    nh3eq_h = (double *)malloc(7425 * sizeof(double));

    tauco_h  = (double *)malloc(7425 * sizeof(double));
    tauco2_h = (double *)malloc(7425 * sizeof(double));
    tauch4_h = (double *)malloc(7425 * sizeof(double));
    tauh2o_h = (double *)malloc(7425 * sizeof(double));
    taunh3_h = (double *)malloc(7425 * sizeof(double));

    P_che_h = (double *)malloc(135 * sizeof(double));
    T_che_h = (double *)malloc(55 * sizeof(double));

    tracer_h = (double *)malloc(nv * point_num * ntr * sizeof(double));

    //  Allocate data in device
    //  Grid
    hipMalloc((void **)&point_local_d, 6 * point_num * sizeof(int));
    hipMalloc((void **)&maps_d, (nl_region + 2) * (nl_region + 2) * nr * sizeof(int));

    //  Operators
    hipMalloc((void **)&nvecoa_d, 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&nvecti_d, 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&nvecte_d, 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&areasT_d, point_num * sizeof(double));
    hipMalloc((void **)&areasTr_d, 6 * point_num * sizeof(double));
    hipMalloc((void **)&func_r_d, 3 * point_num * sizeof(double));
    hipMalloc((void **)&div_d, 7 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&grad_d, 7 * 3 * point_num * sizeof(double));

    //  Altitude (grid)
    hipMalloc((void **)&Altitude_d, nv * sizeof(double));
    hipMalloc((void **)&Altitudeh_d, nvi * sizeof(double));

    //  Longitude-latitude
    hipMalloc((void **)&lonlat_d, 2 * point_num * sizeof(double));

    //  Diagnostics
    hipMalloc((void **)&Mh_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&W_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Wh_d, nvi * point_num * sizeof(double));
    hipMalloc((void **)&Rho_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&pressure_d, nv * point_num * sizeof(double));

    hipMalloc((void **)&tracer_d, nv * point_num * ntr * sizeof(double));
    hipMalloc((void **)&tracers_d, nv * point_num * ntr * sizeof(double));
    hipMalloc((void **)&tracerk_d, nv * point_num * ntr * sizeof(double));

    hipMalloc((void **)&coeq_d, 7425 * sizeof(double));
    hipMalloc((void **)&co2eq_d, 7425 * sizeof(double));
    hipMalloc((void **)&ch4eq_d, 7425 * sizeof(double));
    hipMalloc((void **)&h2oeq_d, 7425 * sizeof(double));
    hipMalloc((void **)&nh3eq_d, 7425 * sizeof(double));

    hipMalloc((void **)&tauco_d, 7425 * sizeof(double));
    hipMalloc((void **)&tauco2_d, 7425 * sizeof(double));
    hipMalloc((void **)&tauch4_d, 7425 * sizeof(double));
    hipMalloc((void **)&tauh2o_d, 7425 * sizeof(double));
    hipMalloc((void **)&taunh3_d, 7425 * sizeof(double));

    hipMalloc((void **)&P_che_d, 135 * sizeof(double));
    hipMalloc((void **)&T_che_d, 55 * sizeof(double));

    //  Temperature
    hipMalloc((void **)&temperature_d, nv * point_num * sizeof(double));

    //  Potential temperature
    hipMalloc((void **)&pt_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&pth_d, nvi * point_num * sizeof(double));

    //  Entalphy
    hipMalloc((void **)&h_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&hh_d, nvi * point_num * sizeof(double));

    //  Advection
    hipMalloc((void **)&Adv_d, nv * point_num * 3 * sizeof(double));

    //  3D vector
    hipMalloc((void **)&v_d, nv * point_num * 3 * sizeof(double));

    //  Effective gravity
    hipMalloc((void **)&gtil_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&gtilh_d, nvi * point_num * sizeof(double));

    //  Slow modes
    hipMalloc((void **)&SlowMh_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&SlowWh_d, nvi * point_num * sizeof(double));
    hipMalloc((void **)&SlowRho_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Slowpressure_d, nv * point_num * sizeof(double));


    //  Deviations
    hipMalloc((void **)&pressures_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Rhos_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Mhs_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&Ws_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Whs_d, nvi * point_num * sizeof(double));


    //  RK-Method
    hipMalloc((void **)&pressurek_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Rhok_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Mhk_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&Wk_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Whk_d, nvi * point_num * sizeof(double));

    //  Vertical integration
    hipMalloc((void **)&Sp_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Sd_d, nv * point_num * sizeof(double));

    //  Diffusion
    hipMalloc((void **)&Kdhz_d, nv * sizeof(double));
    hipMalloc((void **)&Kdh4_d, nv * sizeof(double));
    hipMalloc((void **)&DivM_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&diffpr_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diffmh_d, 3 * nv * point_num * sizeof(double));
    hipMalloc((void **)&diffw_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diffrh_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diff_d, 6 * nv * point_num * sizeof(double));
    hipMalloc((void **)&divg_Mh_d, 3 * nv * point_num * sizeof(double));
    hipMalloc((void **)&difftr_d, nv * point_num * ntr * sizeof(double));

    //  Extras-nan
    hipMalloc((void **)&check_d, sizeof(bool));

    hipMalloc((void **)&vbar_d, 3 * nv * point_num * sizeof(double));
    hipMalloc((void **)&zonal_mean_tab_d, 2 * point_num * sizeof(int));

    if (conservation == true) {
        //  Conservation quantities
        hipMalloc((void **)&Etotal_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&Mass_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&AngMomx_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&AngMomy_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&AngMomz_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&GlobalE_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalMass_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalAMx_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalAMy_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalAMz_d, 1 * sizeof(double));
    }
    // PHY modules
    if (core_benchmark != NO_BENCHMARK)
        phy_modules_init_mem(*this);
}

__host__ bool ESP::initial_values(bool               rest,
                                  const std::string &initial_conditions_filename,
                                  const bool &       continue_sim,
                                  double             timestep_dyn,
                                  XPlanet &          xplanet,
                                  double             kb,
                                  double             mu,
                                  bool               sponge,
                                  bool               DeepModel,
                                  int                TPprof,
                                  int                chemistry,
                                  int &              nstep,
                                  double &           simulation_start_time,
                                  int &              output_file_idx,
                                  bool               conservation) {

    output_file_idx = 0;
    nstep           = 0;

    // Store some general configs
    planet         = xplanet;

    //  Set initial conditions.
    //
    //
    //  Initial atmospheric conditions
    if (rest) {
        double Ha = planet.Rd * planet.Tmean / planet.Gravit;
        for (int i = 0; i < point_num; i++) {
            //
            //          Initial conditions for an isothermal Atmosphere
            //

            for (int lev = 0; lev < nv; lev++) {
                pressure_h[i * nv + lev] = planet.P_Ref * exp(-Altitude_h[lev] / Ha);
                if (TPprof == 0) {
                    temperature_h[i * nv + lev] = planet.Tmean;
                }
                else if (TPprof == 1) {
                    double tau                  = pressure_h[i * nv + lev] / (1e4); //tau = 1 at 0.1 bar
                    double gamma                = 0.6;                              // ratio of sw to lw opacity
                    double f                    = 0.25;
                    temperature_h[i * nv + lev] = pow(3 * planet.Tmean * planet.Tmean * planet.Tmean * planet.Tmean * f * (2 / 3 + 1 / (gamma * sqrt(3)) + (gamma / sqrt(3) - 1 / (gamma * sqrt(3))) * exp(-gamma * tau * sqrt(3))), 0.25);
                }
                if (core_benchmark == HS_DEEP_HOT_JUPITER) {
                    double Ptil = 0.0;
                    if (pressure_h[i * nv + lev] >= 1e5) {
                        Ptil = log10(pressure_h[i * nv + lev] / 100000);
                    }
                    temperature_h[i * nv + lev] = 1696.6986 + 132.2318 * Ptil - 174.30459 * Ptil * Ptil
                                                  + 12.579612 * Ptil * Ptil * Ptil + 59.513639 * Ptil * Ptil * Ptil * Ptil
                                                  + 9.6706522 * Ptil * Ptil * Ptil * Ptil * Ptil
                                                  - 4.1136048 * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil
                                                  - 1.0632301 * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil
                                                  + 0.064400203 * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil
                                                  + 0.035974396 * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil
                                                  + 0.0025740066 * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil * Ptil;
                }
            }

            for (int lev = 0; lev < nv; lev++) {
                //              Density [kg/m3]
                Rho_h[i * nv + lev] = pressure_h[i * nv + lev] / (temperature_h[i * nv + lev] * planet.Rd);

                //              Momentum [kg/m3 m/s]
                Mh_h[i * 3 * nv + 3 * lev + 0] = 0.0;
                Mh_h[i * 3 * nv + 3 * lev + 1] = 0.0;
                Mh_h[i * 3 * nv + 3 * lev + 2] = 0.0;

                //              Vertical momentum [kg/m3 m/s]
                W_h[i * nv + lev]        = 0.0; // Center of the layer.
                Wh_h[i * (nv + 1) + lev] = 0.0; // Layers interface.
            }
            Wh_h[i * (nv + 1) + nv] = 0.0;
        }
        if (core_benchmark == JET_STEADY) {
            //  Number of threads per block.
            const int NTH = 256;

            //  Specify the block sizes.
            dim3 NB((point_num / NTH) + 1, nv, 1);

            hipMemcpy(Altitude_d, Altitude_h, nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(pressure_d, pressure_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(Mh_d, Mh_h, 3 * point_num * nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(Rho_d, Rho_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(temperature_d, temperature_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(lonlat_d, lonlat_h, 2 * point_num * sizeof(double), hipMemcpyHostToDevice);
            setup_jet<<<NB, NTH>>>(Mh_d,
                                   // setup_jet <<< 1, 1 >>>  (Mh_d,
                                   pressure_d,
                                   Rho_d,
                                   temperature_d,
                                   planet.Cp,
                                   planet.Rd,
                                   planet.Omega,
                                   planet.A,
                                   Altitude_d,
                                   lonlat_d,
                                   point_num);

            hipMemcpy(Mh_h, Mh_d, 3 * point_num * nv * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(temperature_h, temperature_d, point_num * nv * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(pressure_h, pressure_d, point_num * nv * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(Rho_h, Rho_d, point_num * nv * sizeof(double), hipMemcpyDeviceToHost);
        }

        simulation_start_time = 0.0;
    }
    else {
        bool load_OK = true;
        // build planet filename
        string planet_filename;

        path   p(initial_conditions_filename);
        int    file_number = 0;
        string basename    = "";

        string parent_path = p.parent();

        if (continue_sim) {
            if (!match_output_file_numbering_scheme(initial_conditions_filename,
                                                    basename,
                                                    file_number)) {
                printf("Loading initial conditions: "
                       "Could not recognise file numbering scheme "
                       "for input %s: (found base: %s, num: %d) \n",
                       initial_conditions_filename.c_str(),
                       basename.c_str(),
                       file_number);
                return false;
            }

            output_file_idx = file_number;

            planet_filename = p.parent() + "/esp_output_planet_" + basename + ".h5";
        }
        else {
            planet_filename = p.parent() + "/" + p.stem() + "_planet.h5";
        }

        // check existence of files
        if (!path_exists(initial_conditions_filename)) {
            printf("initial condition file %s not found.\n", initial_conditions_filename.c_str());
            return false;
        }

        if (!path_exists(planet_filename)) {
            printf("planet_file %s not found.\n", planet_filename.c_str());
            return false;
        }


        printf("Loading planet from: %s\n", planet_filename.c_str());
        printf("Loading initial conditions from: %s\n", initial_conditions_filename.c_str());

        // Check planet data
        {
            // values to check agains variable
            map<string, double> mapValues;

            mapValues["/A"]            = planet.A;
            mapValues["/Top_altitude"] = planet.Top_altitude;
            mapValues["/glevel"]       = glevel;
            mapValues["/vlevel"]       = nv;

            hid_t file_id;
            file_id = H5Fopen(planet_filename.c_str(), H5F_ACC_RDONLY, H5P_DEFAULT);

            bool values_match = true;

            for (const std::pair<std::string, double> &element : mapValues) {
                double value = 0.0;
                load_OK &= load_double_value_from_h5file(file_id, element.first, value);

                if (value != element.second) {
                    printf("mismatch for %s value between config value: %f and initial condition value %f.\n",
                           element.first.c_str(),
                           element.second,
                           value);
                    values_match = false;
                }
            }

            H5Fclose(file_id);

            if (load_OK == false || values_match == false) {
                printf("Could not reload full configuration.\n");

                return false;
            }
        }


        //      Restart from an existing simulation.
        {

            // Load atmospheric data
            hid_t file_id;
            file_id = H5Fopen(initial_conditions_filename.c_str(), H5F_ACC_RDONLY, H5P_DEFAULT);
            // Step number
            load_OK &= load_int_value_from_h5file(file_id, "/nstep", nstep);
            //      Density
            load_OK &= load_double_table_from_h5file(file_id, "/Rho", Rho_h, point_num * nv);

            //      Pressure
            load_OK &= load_double_table_from_h5file(file_id, "/Pressure", pressure_h, point_num * nv);

            //      Horizontal momentum
            load_OK &= load_double_table_from_h5file(file_id, "/Mh", Mh_h, point_num * nv * 3);
            //      Vertical momentum
            load_OK &= load_double_table_from_h5file(file_id, "/Wh", Wh_h, point_num * nvi);

            //      Simulation start time
            load_OK &= load_double_value_from_h5file(file_id, "/simulation_time", simulation_start_time);
            H5Fclose(file_id);
        }


        if (!load_OK)
            return false;

        for (int i = 0; i < point_num; i++)
            for (int lev = 0; lev < nv; lev++)
                temperature_h[i * nv + lev] = pressure_h[i * nv + lev] / (planet.Rd * Rho_h[i * nv + lev]);

        for (int i = 0; i < point_num; i++) {
            for (int lev = 0; lev < nv; lev++) {
                double xi   = Altitude_h[lev];
                double xim1 = Altitudeh_h[lev];
                double xip1 = Altitudeh_h[lev + 1];

                double a = (xi - xip1) / (xim1 - xip1);
                double b = (xi - xim1) / (xip1 - xim1);

                W_h[i * nv + lev] = Wh_h[i * (nv + 1) + lev] * a + Wh_h[i * (nv + 1) + lev + 1] * b;
            }
        }
    }
#ifdef BENCHMARKING
    // recompute temperature from pressure and density, to have correct rounding for binary comparison
    for (int i = 0; i < point_num; i++)
        for (int lev = 0; lev < nv; lev++)
            temperature_h[i * nv + lev] = pressure_h[i * nv + lev] / (planet.Rd * Rho_h[i * nv + lev]);
#endif // BENCHMARKING

    //  Diffusion
    //  Horizontal
    double *Kdhz_h, *Kdh4_h;
    Kdhz_h = new double[nv];
    Kdh4_h = new double[nv];
    for (int lev = 0; lev < nv; lev++) {
        //      Diffusion constant.
        double dbar = sqrt(2 * M_PI / 5) * planet.A / (pow(2, glevel));
        Kdh4_h[lev] = planet.Diffc * pow(dbar, 4.) / timestep_dyn;
        Kdhz_h[lev] = planet.Diffc * pow(dbar, 4.) / timestep_dyn;
    }

    // Input for chemistry
    FILE * infile1;
    int    NT = 55;
    int    NP = 135;
    double dummy;
    if (chemistry == 1) {
        infile1 = fopen("ifile/solar_fEQ_THOR.txt", "r");
        if (infile1 == NULL) {
            printf("\nUnable to open input file.\n");
            exit(EXIT_FAILURE);
        }
        for (int i = 0; i < NT; i++) {
            for (int j = 0; j < NP; j++) {
                if (fscanf(infile1,
                           "%lf %lf %lf %lf %lf %lf %lf",
                           &T_che_h[i],
                           &P_che_h[j],
                           &ch4eq_h[j * NT + i],
                           &coeq_h[j * NT + i],
                           &h2oeq_h[j * NT + i],
                           &co2eq_h[j * NT + i],
                           &nh3eq_h[j * NT + i])
                    != 7) {
                    printf("error parsing ifile/solar_fEQ_THOR.txt\n");
                    fclose(infile1);
                    return false;
                }
            }
        }


        fclose(infile1);

        infile1 = fopen("ifile/solar_chem_time.txt", "r");
        if (infile1 == NULL) {
            printf("\nUnable to open input file.\n");
            return false;
        }
        for (int i = 0; i < NT; i++) {
            for (int j = 0; j < NP; j++) {
                if (fscanf(infile1,
                           "%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf",
                           &T_che_h[i],
                           &P_che_h[j],
                           &tauch4_h[j * NT + i],
                           &tauco_h[j * NT + i],
                           &dummy,
                           &dummy,
                           &tauh2o_h[j * NT + i],
                           &tauco2_h[j * NT + i],
                           &taunh3_h[j * NT + i],
                           &dummy)
                    != 10) {
                    printf("error parsing ifile/solar_chem_time.txt\n");
                    fclose(infile1);
                    return false;
                }
            }
        }

        for (int j = 0; j < NP; j++) P_che_h[j] = log(P_che_h[j]);
        fclose(infile1);

        // CH4
        for (int lev = 0; lev < nv; lev++) {
            for (int i = 0; i < point_num; i++) {
                tracer_h[i * nv * ntr + lev * ntr + 0] = Compute_tracer_host(ch4eq_h,
                                                                             P_che_h,
                                                                             T_che_h,
                                                                             temperature_h[i * nv + lev],
                                                                             pressure_h[i * nv + lev])
                                                         * Rho_h[i * nv + lev];
            }
        }
        // CO
        for (int lev = 0; lev < nv; lev++) {
            for (int i = 0; i < point_num; i++) {
                tracer_h[i * nv * ntr + lev * ntr + 1] = Compute_tracer_host(coeq_h,
                                                                             P_che_h,
                                                                             T_che_h,
                                                                             temperature_h[i * nv + lev],
                                                                             pressure_h[i * nv + lev])
                                                         * Rho_h[i * nv + lev];
            }
        }
        // H2O
        for (int lev = 0; lev < nv; lev++) {
            for (int i = 0; i < point_num; i++) {
                tracer_h[i * nv * ntr + lev * ntr + 2] = Compute_tracer_host(h2oeq_h,
                                                                             P_che_h,
                                                                             T_che_h,
                                                                             temperature_h[i * nv + lev],
                                                                             pressure_h[i * nv + lev])
                                                         * Rho_h[i * nv + lev];
            }
        }
        // CO2
        for (int lev = 0; lev < nv; lev++) {
            for (int i = 0; i < point_num; i++) {
                tracer_h[i * nv * ntr + lev * ntr + 3] = Compute_tracer_host(co2eq_h,
                                                                             P_che_h,
                                                                             T_che_h,
                                                                             temperature_h[i * nv + lev],
                                                                             pressure_h[i * nv + lev])
                                                         * Rho_h[i * nv + lev];
            }
        }
        // NH3
        for (int lev = 0; lev < nv; lev++) {
            for (int i = 0; i < point_num; i++) {
                tracer_h[i * nv * ntr + lev * ntr + 4] = Compute_tracer_host(nh3eq_h,
                                                                             P_che_h,
                                                                             T_che_h,
                                                                             temperature_h[i * nv + lev],
                                                                             pressure_h[i * nv + lev])
                                                         * Rho_h[i * nv + lev];
            }
        }
    }

    //  Copy memory to the devide
    hipMemcpy(point_local_d, point_local_h, 6 * point_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(maps_d, maps_h, (nl_region + 2) * (nl_region + 2) * nr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Altitude_d, Altitude_h, nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Altitudeh_d, Altitudeh_h, nvi * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecoa_d, nvecoa_h, 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecti_d, nvecti_h, 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecte_d, nvecte_h, 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(areasTr_d, areasTr_h, 6 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(areasT_d, areasT_h, point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(lonlat_d, lonlat_h, 2 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(func_r_d, func_r_h, 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temperature_d, temperature_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Mh_d, Mh_h, point_num * nv * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(W_d, W_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Wh_d, Wh_h, point_num * nvi * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Rho_d, Rho_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pressure_d, pressure_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(div_d, div_h, 7 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(grad_d, grad_h, 7 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kdhz_d, Kdhz_h, nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kdh4_d, Kdh4_h, nv * sizeof(double), hipMemcpyHostToDevice);

    if (sponge == true)
        hipMemcpy(zonal_mean_tab_d, zonal_mean_tab_h, 2 * point_num * sizeof(int), hipMemcpyHostToDevice);

    if (chemistry == 1) {
        hipMemcpy(coeq_d, coeq_h, 7425 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(ch4eq_d, ch4eq_h, 7425 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(h2oeq_d, h2oeq_h, 7425 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(co2eq_d, co2eq_h, 7425 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(nh3eq_d, nh3eq_h, 7425 * sizeof(double), hipMemcpyHostToDevice);

        hipMemcpy(tauco_d, tauco_h, 7425 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(tauch4_d, tauch4_h, 7425 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(tauh2o_d, tauh2o_h, 7425 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(tauco2_d, tauco2_h, 7425 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(taunh3_d, taunh3_h, 7425 * sizeof(double), hipMemcpyHostToDevice);

        hipMemcpy(P_che_d, P_che_h, 135 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(T_che_d, T_che_h, 55 * sizeof(double), hipMemcpyHostToDevice);

        hipMemcpy(tracer_d, tracer_h, point_num * nv * ntr * sizeof(double), hipMemcpyHostToDevice);
        hipMemset(tracers_d, 0, sizeof(double) * nv * point_num * ntr);
        hipMemset(tracerk_d, 0, sizeof(double) * nv * point_num * ntr);
    }

    //  Initialize arrays
    hipMemset(Adv_d, 0, sizeof(double) * 3 * point_num * nv);
    hipMemset(v_d, 0, sizeof(double) * nv * point_num * 3);
    hipMemset(pt_d, 0, sizeof(double) * nv * point_num);
    hipMemset(pth_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(SlowMh_d, 0, sizeof(double) * nv * point_num * 3);
    hipMemset(SlowWh_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(SlowRho_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Slowpressure_d, 0, sizeof(double) * nv * point_num);
    hipMemset(h_d, 0, sizeof(double) * nv * point_num);
    hipMemset(hh_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(Rhos_d, 0, sizeof(double) * nv * point_num);
    hipMemset(pressures_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Mhs_d, 0, sizeof(double) * nv * point_num * 3);
    hipMemset(Ws_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Whs_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(gtil_d, 0, sizeof(double) * nv * point_num);
    hipMemset(gtilh_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(Rhok_d, 0, sizeof(double) * nv * point_num);
    hipMemset(pressurek_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Mhk_d, 0, sizeof(double) * nv * point_num * 3);
    hipMemset(Wk_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Whk_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(Sp_d, 0, sizeof(double) * point_num * nv);
    hipMemset(Sd_d, 0, sizeof(double) * point_num * nv);
    hipMemset(DivM_d, 0, sizeof(double) * point_num * 3 * nv);
    hipMemset(diffpr_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diffmh_d, 0, sizeof(double) * 3 * nv * point_num);
    hipMemset(diffw_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diffrh_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diff_d, 0, sizeof(double) * 6 * nv * point_num);
    hipMemset(divg_Mh_d, 0, sizeof(double) * 3 * nv * point_num);
    hipMemset(difftr_d, 0, sizeof(double) * nv * point_num * ntr);

    delete[] Kdh4_h;
    delete[] Kdhz_h;

    return true;
}

__host__ ESP::~ESP() {

    //
    //  Description: Frees the memory space.
    //
    //  Host
    free(point_local_h);
    free(maps_h);
    free(lonlat_h);
    free(Altitude_h);
    free(Altitudeh_h);
    free(nvecoa_h);
    free(nvecti_h);
    free(nvecte_h);
    free(areasTr_h);
    free(div_h);
    free(grad_h);
    free(func_r_h);
    free(Rho_h);
    free(pressure_h);
    free(temperature_h);
    free(Mh_h);
    free(W_h);
    free(Wh_h);

    free(tauch4_h);
    free(tauco_h);
    free(tauh2o_h);
    free(tauco2_h);
    free(taunh3_h);

    free(ch4eq_h);
    free(coeq_h);
    free(h2oeq_h);
    free(co2eq_h);
    free(nh3eq_h);

    free(P_che_h);
    free(T_che_h);

    //  Device
    hipFree(point_local_d);
    hipFree(maps_d);
    hipFree(Altitude_d);
    hipFree(Altitudeh_d);
    hipFree(nvecoa_d);
    hipFree(nvecti_d);
    hipFree(nvecte_d);
    hipFree(areasT_d);
    hipFree(areasTr_d);
    hipFree(lonlat_d);
    hipFree(div_d);
    hipFree(grad_d);
    hipFree(func_r_d);
    hipFree(Rho_d);
    hipFree(pressure_d);
    hipFree(temperature_d);
    hipFree(W_d);
    hipFree(Wh_d);
    hipFree(h_d);
    hipFree(hh_d);
    hipFree(Adv_d);
    hipFree(gtil_d);
    hipFree(gtilh_d);
    hipFree(v_d);
    hipFree(pt_d);
    hipFree(pth_d);
    hipFree(SlowMh_d);
    hipFree(SlowWh_d);
    hipFree(SlowRho_d);
    hipFree(Slowpressure_d);
    hipFree(Rhok_d);
    hipFree(pressurek_d);
    hipFree(Mhk_d);
    hipFree(Whk_d);
    hipFree(Wk_d);
    hipFree(Rhos_d);
    hipFree(pressures_d);
    hipFree(Mhs_d);
    hipFree(Whs_d);
    hipFree(Ws_d);

    hipFree(ch4eq_d);
    hipFree(coeq_d);
    hipFree(h2oeq_d);
    hipFree(co2eq_d);
    hipFree(nh3eq_d);

    hipFree(tauch4_d);
    hipFree(tauco_d);
    hipFree(tauh2o_d);
    hipFree(tauco2_d);
    hipFree(taunh3_d);

    hipFree(tracer_d);
    hipFree(tracers_d);
    hipFree(tracerk_d);

    hipFree(P_che_d);
    hipFree(T_che_d);

    hipFree(Sd_d);
    hipFree(Sp_d);
    hipFree(Kdhz_d);
    hipFree(Kdh4_d);
    hipFree(DivM_d);
    hipFree(diffpr_d);
    hipFree(diffmh_d);
    hipFree(diffw_d);
    hipFree(diffrh_d);
    hipFree(diff_d);
    hipFree(difftr_d);
    hipFree(divg_Mh_d);

    //  Conservation quantities
    hipFree(Etotal_d);
    hipFree(Mass_d);
    hipFree(AngMomx_d);
    hipFree(AngMomy_d);
    hipFree(AngMomz_d);
    hipFree(GlobalE_d);
    hipFree(GlobalMass_d);
    hipFree(GlobalAMx_d);
    hipFree(GlobalAMy_d);
    hipFree(GlobalAMz_d);

    if (core_benchmark != NO_BENCHMARK)
        phy_modules_free_mem();


    printf("\n\n Free memory!\n\n");
}
